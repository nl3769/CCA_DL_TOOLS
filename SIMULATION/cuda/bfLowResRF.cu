
#include <hip/hip_runtime.h>
extern "C" { // ---> [kernel]

/* -------------------------------------------------------------------------------------------------------------------- */
/* --------------------------------------- DAS LOW RESOLUTION  -------------------------------------------------------- */
/* ------------------------------------------ LAINE NOLANN ------------------------------------------------------------ */
/* -------------------------------------------------------------------------------------------------------------------- */

  __global__ void das_low_res(double* I, double* RF, double* pos_el_x, double* pos_el_z, int nb_el, int nb_sample, int W, int H, double c, double fs, double* apod, int id_tx, double* x_img, double* z_img, double t_offset)
  {

    // --- variables
    int idx;            // index of the image
    double delay;       // exact time (to read RF signals)
    int id_t;           // time index used for interpolation
    double tof;         // time of flight
    int id_apod_tx;     // tx apodization
    int id_apod_rx;     // rx apodization
    double rx_dst;      // -> todo
    double tx_dst;      // -> todo
    double apod_tx;     // -> todo
    double apod_rx;     // -> todo
    double val_sample;  // -> todo

    int col=blockIdx.x * blockDim.x + threadIdx.x;  // column in the grid image
    int row=blockIdx.y * blockDim.y + threadIdx.y;  // row in the grid image

    if (col < W  && row < H)
    {
        idx = col * H + row;
        id_apod_tx = W*H*id_tx + idx;
        tx_dst = sqrt( (pos_el_x[id_tx] - x_img[col])*(pos_el_x[id_tx] - x_img[col]) + (pos_el_z[id_tx] - z_img[row])*(pos_el_z[id_tx] - z_img[row]) );
        // --- loop over rx elements
        for (int id_rx=0; id_rx < nb_el; id_rx++)
        {
            rx_dst=sqrt( (pos_el_x[id_rx] - x_img[col])*(pos_el_x[id_rx] - x_img[col]) + (pos_el_z[id_rx] - z_img[row])*(pos_el_z[id_rx] - z_img[row]) );
            tof=(rx_dst + tx_dst)/c  + t_offset;
            delay=tof * fs + 1;
            if(delay>=1 && delay<=(nb_sample-1))
            {
              id_t=(int)floor(delay);
              id_apod_rx = W*H*id_rx + idx;
              val_sample=RF[nb_sample * id_rx + id_t] * (id_t + 1 - delay) + RF[nb_sample * id_rx + id_t + 1] * (delay - id_t);
              apod_tx=apod[id_apod_tx];
              apod_rx=apod[id_apod_rx];
              I[idx] += apod_tx * apod_rx * val_sample;
            }
        }
    }
  }
}
